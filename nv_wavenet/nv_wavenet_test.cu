/******************************************************************************
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "matrix.h"
#include "nv_wavenet.cuh"
#include "nv_wavenet_util.cuh"
#include "nv_wavenet_reference.h"
#include <assert.h>
#include <stdio.h>
#include <vector>

Matrix* createMatrix(int r, int c) {
    float mean = 0.0;
    float scale = 0.5 / r;
    Matrix* m = new Matrix(r,c,false);
    m->randomize(mean,scale);
   return m;
}

template <typename T_weight, typename T_data, int R, int S, int A>
void runTest(int num_layers, int max_dilation, int batch_size, int num_iterations, int samples_per_iteration, int impl, bool inputsFromDevice=false, bool weightsFromDevice=false) {

    float mean = 0.0;
    float scale = 0.5 / R;

    // Just encode one-hot vector as an integer
    std::vector<int> yInPrev(batch_size);
    std::vector<int> yInCur(batch_size);

    for (int b=0; b<batch_size; b++) {
        yInPrev[b] = rand() % A;
        yInCur[b] = rand() % A;
    }
    std::vector<int> yOut(batch_size);

    Matrix outputSelectors(batch_size,samples_per_iteration);
    outputSelectors.randomize(0.5,1.0);

    Matrix embeddingsPrev(R,A,false);
    Matrix embeddingsCur(R,A,false);

    embeddingsPrev.randomize(mean,scale);
    embeddingsCur.randomize(mean,scale);

    std::vector<Matrix*> Wprev(num_layers);
    std::vector<Matrix*> Wcur(num_layers);
    std::vector<Matrix*> Bh(num_layers);
    std::vector<Matrix*> Wres(num_layers);
    std::vector<Matrix*> Bres(num_layers);
    std::vector<Matrix*> Wskip(num_layers);
    std::vector<Matrix*> Bskip(num_layers);
    std::vector<Matrix*> skipOut(num_layers+1);

    // Retain results for dilated inputs
    std::vector<std::vector<Matrix*>> Xt(samples_per_iteration);
    for (int sample=0; sample<samples_per_iteration; sample++) {
        Xt[sample].resize(num_layers+1);
    }

    for (int l=0; l<num_layers; l++) {
        // Weights
        Wprev[l] = createMatrix(2*R,R);
        Wcur[l] = createMatrix(2*R,R);
        Bh[l] = createMatrix(2*R,1);
        Wres[l] = createMatrix(R,R);
        Bres[l] = createMatrix(R,1);
        Wskip[l] = createMatrix(S,R);
        Bskip[l] = createMatrix(S,1);

        // Activations
        skipOut[l] = createMatrix(S,batch_size);
    }

    for (int sample=0; sample<samples_per_iteration; sample++) {
        for (int layer=0; layer<num_layers+1; layer++) {
            Xt[sample][layer] = createMatrix(R, batch_size);
        }
    }

    Matrix WskipOut(A,S,false);
    WskipOut.randomize(mean,scale);
    Matrix BskipOut(A,1,false);
    BskipOut.randomize(mean, scale);
    Matrix Wout(A,A,false);
    Wout.randomize(mean,scale);
    Matrix Bout(A,1,false);
    Bout.randomize(mean,scale);

    Matrix skipOutFinal(A,batch_size,false);
    Matrix out(A,batch_size,false);
    Matrix p(A,batch_size,false);

    Matrix zero(S,batch_size,false);
    for (int row = 0; row < S; row++) {
        for (int col = 0; col < batch_size; col++) {
            zero.set(row,col,0.f);
        }
    }

    nvWavenetReference ref(num_layers, batch_size, samples_per_iteration, R, S, A, max_dilation);
    nvWavenetInfer<T_weight,T_data,R,S,A>* infer = new nvWavenetInfer<T_weight,T_data,R,S,A>(num_layers, max_dilation, batch_size, samples_per_iteration, impl);

    ref.setEmbeddings(embeddingsPrev.data(), embeddingsCur.data());
    for (int l=0; l<num_layers; l++) {
        ref.setLayerWeights(l, Wprev[l]->data(), Wcur[l]->data(), Bh[l]->data(), Wres[l]->data(), Bres[l]->data(), Wskip[l]->data(), Bskip[l]->data());
    }
    ref.setOutWeights(WskipOut.data(), BskipOut.data(), Wout.data(), Bout.data());

    if (weightsFromDevice) {
        float* d_embeddingsPrev;
        float* d_embeddingsCur;
        gpuErrChk(hipMalloc(&d_embeddingsPrev, R*A*sizeof(float)));
        gpuErrChk(hipMemcpy(d_embeddingsPrev, embeddingsPrev.data(), R*A*sizeof(float), hipMemcpyHostToDevice));
        gpuErrChk(hipMalloc(&d_embeddingsCur, R*A*sizeof(float)));
        gpuErrChk(hipMemcpy(d_embeddingsCur, embeddingsCur.data(), R*A*sizeof(float), hipMemcpyHostToDevice));

        infer->setEmbeddings(d_embeddingsPrev, d_embeddingsCur);

        gpuErrChk(hipFree(d_embeddingsPrev));
        gpuErrChk(hipFree(d_embeddingsCur));

        float* d_Wprev;
        float* d_Wcur;
        float* d_Bh;
        float* d_Wres;
        float* d_Bres;
        float* d_Wskip;
        float* d_Bskip;
        for (int l=0; l<num_layers; l++) {
            gpuErrChk(hipMalloc(&d_Wprev, 2*R*R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Wprev, Wprev[l]->data(), 2*R*R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Wcur, 2*R*R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Wcur, Wcur[l]->data(), 2*R*R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Bh, 2*R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Bh, Bh[l]->data(), 2*R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Wres, R*R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Wres, Wres[l]->data(), R*R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Bres, R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Bres, Bres[l]->data(), R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Wskip, S*R*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Wskip, Wskip[l]->data(), S*R*sizeof(float), hipMemcpyHostToDevice));
            gpuErrChk(hipMalloc(&d_Bskip, S*sizeof(float)));
            gpuErrChk(hipMemcpy(d_Bskip, Bskip[l]->data(), S*sizeof(float), hipMemcpyHostToDevice));

            infer->setLayerWeights(l, d_Wprev, d_Wcur, d_Bh, d_Wres, d_Bres, d_Wskip, d_Bskip);

            gpuErrChk(hipFree(d_Wprev));
            gpuErrChk(hipFree(d_Wcur));
            gpuErrChk(hipFree(d_Bh));
            gpuErrChk(hipFree(d_Wres));
            gpuErrChk(hipFree(d_Bres));
            gpuErrChk(hipFree(d_Wskip));
            gpuErrChk(hipFree(d_Bskip));
        }

        float* d_WskipOut;
        float* d_BskipOut;
        float* d_Wout;
        float* d_Bout;

        gpuErrChk(hipMalloc(&d_WskipOut, A*S*sizeof(float)));
        gpuErrChk(hipMemcpy(d_WskipOut, WskipOut.data(), A*S*sizeof(float), hipMemcpyHostToDevice));
        gpuErrChk(hipMalloc(&d_BskipOut, A*sizeof(float)));
        gpuErrChk(hipMemcpy(d_BskipOut, BskipOut.data(), A*sizeof(float), hipMemcpyHostToDevice));
        gpuErrChk(hipMalloc(&d_Wout, A*A*sizeof(float)));
        gpuErrChk(hipMemcpy(d_Wout, Wout.data(), A*A*sizeof(float), hipMemcpyHostToDevice));
        gpuErrChk(hipMalloc(&d_Bout, A*sizeof(float)));
        gpuErrChk(hipMemcpy(d_Bout, Bout.data(), A*sizeof(float), hipMemcpyHostToDevice));
        
        infer->setOutWeights(d_WskipOut, d_BskipOut, d_Wout, d_Bout);

        gpuErrChk(hipFree(d_WskipOut));
        gpuErrChk(hipFree(d_BskipOut));
        gpuErrChk(hipFree(d_Wout));
        gpuErrChk(hipFree(d_Bout));
        
    }
    else {
        infer->setEmbeddings(embeddingsPrev.data(), embeddingsCur.data());
        for (int l=0; l<num_layers; l++) {
            infer->setLayerWeights(l, Wprev[l]->data(), Wcur[l]->data(), Bh[l]->data(), Wres[l]->data(), Bres[l]->data(), Wskip[l]->data(), Bskip[l]->data());
        }
        infer->setOutWeights(WskipOut.data(), BskipOut.data(), Wout.data(), Bout.data());
    }

    Matrix zeroMatrix(R,batch_size,false);
    for (int row=0; row<R; row++) {
        for (int col=0; col<batch_size; col++) {
            zeroMatrix.set(row,col,0.f);
        }
    }

    Matrix Lh(2*R,samples_per_iteration*num_layers*batch_size);
    assert(Lh.data());
    Lh.randomize(mean,scale);

    ref.setInputs(Lh.data(), outputSelectors.data());

    if (inputsFromDevice) {
        float* d_Lh;
        gpuErrChk(hipMalloc(&d_Lh, 2*R*samples_per_iteration*num_layers*batch_size*sizeof(float)));
        float* d_outputSelectors;
        gpuErrChk(hipMalloc(&d_outputSelectors,samples_per_iteration*batch_size*sizeof(float)));

        gpuErrChk(hipMemcpy(d_Lh, Lh.data(), 2*R*samples_per_iteration*num_layers*batch_size*sizeof(float), hipMemcpyHostToDevice));
        gpuErrChk(hipMemcpy(d_outputSelectors, outputSelectors.data(), samples_per_iteration*batch_size*sizeof(float), hipMemcpyHostToDevice));

        infer->setInputs(d_Lh, d_outputSelectors);

        gpuErrChk(hipFree(d_Lh));
        gpuErrChk(hipFree(d_outputSelectors));
    }
    else {
        infer->setInputs(Lh.data(), outputSelectors.data());
    }

    for (int i=0; i<num_iterations; i++) {
        printf("Iteration: %d\n", i);

        // Run reference implementation


        int batch_size_per_block = ((batch_size % 4) == 0) ? 4 : ((batch_size % 2) == 0) ? 2 : 1;

        int* refYout = (int*)malloc(samples_per_iteration*batch_size*sizeof(int));
        int* mcYout = (int*)malloc(samples_per_iteration*batch_size*sizeof(int));

        ref.run(samples_per_iteration, batch_size, refYout);

        assert(infer->run_chunks(7, [](int*, int, int){}, samples_per_iteration, batch_size, mcYout, batch_size_per_block));
        gpuErrChk(hipDeviceSynchronize());

        // Check results

        for (int l=0; l<num_layers; l++) {

            printf("Checking layer %d\n", l);

            Matrix refXout(R,batch_size);
            Matrix refSkipOut(S, batch_size);
            ref.getXtOut(l, refXout.data());
            ref.getSkipOut(l, refSkipOut.data());

            Matrix mcXout(R,batch_size,false);
            Matrix mcSkipOut(S,batch_size,false);
            infer->getXtOut(l, mcXout.data());
            infer->getSkipOut(l, mcSkipOut.data());

            matrix_compare("Xout", refXout, mcXout, 1.e-2);
            matrix_compare("skipOut", refSkipOut, mcSkipOut, 1.e-2, true);
        }

        Matrix refSkipOutFinal(A,batch_size);
        ref.getZs(refSkipOutFinal.data());

        Matrix mcSkipOutFinal(A,batch_size,false);
        infer->getZs(mcSkipOutFinal.data());

        matrix_compare("Zs", refSkipOutFinal, mcSkipOutFinal, 1.e-4, true);

        Matrix refOut(A,batch_size);
        ref.getZa(refOut.data());

        Matrix mcOut(A,batch_size,false);
        infer->getZa(mcOut.data());

        matrix_compare("Za", refOut, mcOut, 1.e-4);

        Matrix refP(A,batch_size);
        ref.getP(refP.data());

        Matrix mcP(A,batch_size,false);
        infer->getP(mcP.data());
        matrix_compare("p",refP,mcP,1.e-3);

        printf("Comparing yOut\n");

        for (int i=0; i<samples_per_iteration*batch_size; i++) {
            assert(refYout[i] == mcYout[i]);
        }
        free(mcYout);
        free(refYout);

        printf("SUCCESS!\n");
    }


    // Clean up

    delete infer;

    for (int l=0; l<num_layers; l++) {
        delete Wprev[l];
        delete Wcur[l];
        delete Bh[l];
        delete Wres[l];
        delete Bres[l];
        delete Wskip[l];
        delete Bskip[l];
        for (int sample=0; sample<samples_per_iteration;sample++) {
            delete Xt[sample][l];
        }
        delete skipOut[l];
    }
}

int main(int argc, char* argv[]) {

    int num_layers = 20;
    int batch_size = 16;

    if (argc > 1) num_layers = atoi(argv[1]);
    if (argc > 2) batch_size  = atoi(argv[2]);

    // How many samples to generate each time we invoke the kernel
    const int SAMPLES_PER_ITERATION = 8;
    const int MAX_DILATION = SAMPLES_PER_ITERATION;

    srand(3);

    printf("Testing R=32, S=128\n");
    printf("   Testing Single-Block\n");
    runTest<float,float,32,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 1);
    printf("   Testing Dual-Block\n");
    runTest<float,float,32,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 2);
    printf("   Testing Persistent\n");
    runTest<float,float,32,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3);
    printf("   Testing Manyblock\n");
    runTest<float,float,32,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 4);

    srand(10);

    printf("Testing R=64, S=128\n");
    printf("   Testing Single-Block\n");
    runTest<float,float,64,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 1, true, false);
    printf("   Testing Dual-Block\n");
    runTest<float,float,64,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 2, false, true);
    printf("   Testing Persistent\n");
    runTest<float,float,64,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3, true, true);
    printf("   Testing Manyblock\n");
    runTest<float,float,64,128, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 4, true, true);

    srand(30);

    printf("Testing R=64, S=256\n");
    printf("    Testing Single-Block\n");
    runTest<float,float,64,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 1);
    printf("    Testing Dual-Block\n");
    runTest<float,float,64,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 2);
    printf("    Testing Persistent\n");
    runTest<float,float,64,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3);
    printf("   Testing Manyblock\n");
    runTest<float,float,64,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 4);

    srand(50);

    printf("Testing R=128, S=256\n");
    printf("    Testing Persistent\n");
    runTest<float,float,128,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3);
    printf("   Testing Manyblock\n");
    runTest<float,float,128,256, 256>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 4);

    srand(70);

    printf("Testing A=512\n");
    printf("    Testing Persistent\n");
    runTest<float,float,64,128, 512>(num_layers, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3);
    printf("Testing A=1024\n");
    printf("    Testing Persistent\n");
    runTest<float,float,128,256, 1024>(12, MAX_DILATION, batch_size, 2, SAMPLES_PER_ITERATION, 3);
}
